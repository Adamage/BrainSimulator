#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif


#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <hip/hip_complex.h>

extern "C"  
{	
	//kernel code
	__global__ void InvertValuesKernel(float *input, float* outputs, int size)
	{		
		int id = blockDim.x * blockIdx.y * gridDim.x	
			+ blockDim.x*blockIdx.x				
			+ threadIdx.x;

		if(id < size) 
		{
			outputs[id] = 1.00f - input[id];
		}		
	}

	__global__ void InvertLengthComplexKernel(hipFloatComplex* input, hipFloatComplex* outputs, int size)
	{
		int id = blockDim.x*blockIdx.y*gridDim.x	
			+ blockDim.x*blockIdx.x				
			+ threadIdx.x;

		if (id >= size)
			return;


		hipFloatComplex val = input[id];
		float length = hipCabsf(val);

		if (length < 0.00001f)
			length = 0;
		else
			length = 1 / length;

		length *= length;

		val.x *= length;
		val.y *= length;

		outputs[id] = val;
	}

	__global__ void InvertPermutationKernel(float* input, float* output, int size)
	{
		int id = blockDim.x*blockIdx.y*gridDim.x	
			+ blockDim.x*blockIdx.x				
			+ threadIdx.x;

		if (id >= size)
			return;


		int temp = __float2int_rn(input[id]);

		if (input == output)
			__syncthreads();

		output[temp] = id;
	}
}
