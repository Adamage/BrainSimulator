#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

// Gaussian regularization coefficient
__constant__ float RegularizationCoefficient;

extern "C"
{
	__global__ void L1TermKernel(
		float *weightPtr,
		float *L1TermPtr,
		int weights
		)
	{
		extern __shared__ float partialSum[];

		unsigned int blockSize = blockDim.x;
		unsigned int tid = threadIdx.x;
		unsigned int idx = tid;

		partialSum[tid] = 0;
		while (idx < weights) { partialSum[tid] += abs(weightPtr[idx]); idx += blockSize; }

		if (blockSize >= 1024) { if (tid < 512) { partialSum[tid] += partialSum[tid + 512]; } __syncthreads(); }
		if (blockSize >= 512) { if (tid < 256) { partialSum[tid] += partialSum[tid + 256]; } __syncthreads(); }
		if (blockSize >= 256) { if (tid < 128) { partialSum[tid] += partialSum[tid + 128]; } __syncthreads(); }
		if (blockSize >= 128) { if (tid < 64) { partialSum[tid] += partialSum[tid + 64]; } __syncthreads(); }
		if (tid < 32) {
			if (blockSize >= 64) partialSum[tid] += partialSum[tid + 32];
			if (blockSize >= 32) partialSum[tid] += partialSum[tid + 16];
			if (blockSize >= 16) partialSum[tid] += partialSum[tid + 8];
			if (blockSize >= 8) partialSum[tid] += partialSum[tid + 4];
			if (blockSize >= 4) partialSum[tid] += partialSum[tid + 2];
			if (blockSize >= 2) partialSum[tid] += partialSum[tid + 1];
		}
		if (tid == 0)
			*L1TermPtr = partialSum[0];
	}

	__global__ void L2TermKernel(
		float *weightPtr,
		float *L2TermPtr,
		int weights
		)
	{
		extern __shared__ float partialSum[];

		unsigned int blockSize = blockDim.x;
		unsigned int tid = threadIdx.x;
		unsigned int idx = tid;

		partialSum[tid] = 0;
		while (idx < weights) { partialSum[tid] += weightPtr[idx] * weightPtr[idx]; idx += blockSize; }

		if (blockSize >= 1024) { if (tid < 512) { partialSum[tid] += partialSum[tid + 512]; } __syncthreads(); }
		if (blockSize >= 512) { if (tid < 256) { partialSum[tid] += partialSum[tid + 256]; } __syncthreads(); }
		if (blockSize >= 256) { if (tid < 128) { partialSum[tid] += partialSum[tid + 128]; } __syncthreads(); }
		if (blockSize >= 128) { if (tid < 64) { partialSum[tid] += partialSum[tid + 64]; } __syncthreads(); }
		if (tid < 32) {
			if (blockSize >= 64) partialSum[tid] += partialSum[tid + 32];
			if (blockSize >= 32) partialSum[tid] += partialSum[tid + 16];
			if (blockSize >= 16) partialSum[tid] += partialSum[tid + 8];
			if (blockSize >= 8) partialSum[tid] += partialSum[tid + 4];
			if (blockSize >= 4) partialSum[tid] += partialSum[tid + 2];
			if (blockSize >= 2) partialSum[tid] += partialSum[tid + 1];
		}
		if (tid == 0)
			*L2TermPtr = 0.5f * partialSum[0];
	}

	__global__ void GaussianRegularizationKernel(
		float *inputPtr,
		int prevLayerSize,
		float *regularizationPtr
		)
	{
		extern __shared__ float partialSum[];

		unsigned int blockSize = blockDim.x;
		unsigned int tid = threadIdx.x;
		unsigned int idx = tid;

		partialSum[tid] = 0;
		while (idx < prevLayerSize / 2)
		{
			float mu_sq = pow(inputPtr[idx], 2);
			float sigma_sq = pow(inputPtr[idx + prevLayerSize / 2], 2);
			partialSum[tid] += mu_sq + sigma_sq - log(sigma_sq);
			idx += blockSize;
		}

		if (blockSize >= 1024) { if (tid < 512) { partialSum[tid] += partialSum[tid + 512]; } __syncthreads(); }
		if (blockSize >= 512) { if (tid < 256) { partialSum[tid] += partialSum[tid + 256]; } __syncthreads(); }
		if (blockSize >= 256) { if (tid < 128) { partialSum[tid] += partialSum[tid + 128]; } __syncthreads(); }
		if (blockSize >= 128) { if (tid < 64) { partialSum[tid] += partialSum[tid + 64]; } __syncthreads(); }
		if (tid < 32) {
			if (blockSize >= 64) partialSum[tid] += partialSum[tid + 32];
			if (blockSize >= 32) partialSum[tid] += partialSum[tid + 16];
			if (blockSize >= 16) partialSum[tid] += partialSum[tid + 8];
			if (blockSize >= 8) partialSum[tid] += partialSum[tid + 4];
			if (blockSize >= 4) partialSum[tid] += partialSum[tid + 2];
			if (blockSize >= 2) partialSum[tid] += partialSum[tid + 1];
		}
		if (tid == 0)
			*regularizationPtr = partialSum[0];
	}

    __global__ void GaussianRegularizationDeltaKernel(
            float* prevLayerOutputPtr,
            int prevLayerOutputCount,
            float* prevLayerInputPtr,
            int prevLayerInputCount,
            float* prevLayerWeights,
            float* prevPrevLayerDelta
            )
    {
            // i: previous layer output (which is mu, sigma params)
            int i = blockDim.x * blockIdx.y * gridDim.x     //rows preceeding current row in grid
                    + blockDim.x * blockIdx.x                               //blocks preceeding current block
                    + threadIdx.x;
 
            if (i < prevLayerOutputCount / 2)
            {
                    // first half are mu params
                    for (int j = 0; j < prevLayerInputCount; j++)
                    {
                            float w = prevLayerWeights[j * prevLayerOutputCount];
                            float x_sq = pow(prevLayerInputPtr[j], 2);
                            prevPrevLayerDelta[j] += RegularizationCoefficient * w * x_sq;
                    }
            }
            else if (i < prevLayerOutputCount)
            {
                    // second half are sigma params
                    for (int j = 0; j < prevLayerInputCount; j++)
                    {
                            float w = prevLayerWeights[j * prevLayerOutputCount];
                            float x_sq = pow(prevLayerInputPtr[j], 2);
                            prevPrevLayerDelta[j] += RegularizationCoefficient * (w * x_sq - 1 / w);
                    }
            }
    }

	__global__ void DropoutMaskKernel(
		float *dropoutMaskPtr,
		float dropout,
		int inputSize
		)
	{
		int i = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (i < inputSize)
		{
			dropoutMaskPtr[i] = dropout > dropoutMaskPtr[i];
			/*if (dropoutMaskPtr[i] > dropout)
				dropoutMaskPtr[i] = 0.0f;
			else
				dropoutMaskPtr[i] = 1.0f;*/
		}
	}
}