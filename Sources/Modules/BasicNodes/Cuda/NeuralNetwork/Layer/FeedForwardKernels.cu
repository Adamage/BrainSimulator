#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

#include "..\Activation\ActivationFunction.cu"

extern "C"
{
	__global__ void FullyConnectedForwardKernel(
		ActivationFunctionEnum activationFunction,
		float *inputPtr,
		float *outputPtr,
		float *weightPtr,
		float *neuronInputPtr,
		float *biasPtr,
		float *dropoutMaskPtr,
		float dropout,
		int prevLayerSize,
		int thisLayerSize
		)
	{
		// i: prev. layer neuron id
		// j: current layer neuron id
		int i;
		int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (j < thisLayerSize)
		{
			if (dropoutMaskPtr[j])
			{
				neuronInputPtr[j] = 0;
				outputPtr[j] = 0;
			}
			else
			{
				float sum = 0.0;
				int index = j;
				for (i = 0; i < prevLayerSize; i++) {
					sum += weightPtr[index] * inputPtr[i];
					index += thisLayerSize;
				}
				// add bias
				sum += biasPtr[j];

				// sum neuron input
				neuronInputPtr[j] = sum;

				// set output value
				outputPtr[j] = Evaluate(activationFunction, sum) / (1.0f - dropout);
			}
		}
	}

	__global__ void OneToOneForwardKernel(
		ActivationFunctionEnum activationFunction,
		float *inputPtr,
		float *outputPtr,
		int layerSize
		)
	{
		// i: neuron id
		int i = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (i < layerSize)
			outputPtr[i] = Evaluate(activationFunction, inputPtr[i]);
	}


    __global__ void GaussianForwardSamplingKernel(
		float* gaussianParamsInputPtr,
		float* outputPtr,
		float* biasPtr,
		float* randomNormalPtr,
		int prevLayerSize,
		int thisLayerSize
		)
	{
		// j: current layer neuron id
		int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (j < thisLayerSize)
		{
			float mu = gaussianParamsInputPtr[j];
			float sigma = gaussianParamsInputPtr[j + prevLayerSize / 2];
			float x = randomNormalPtr[j];
				
			// sample Gaussian from Uniform
			//float t = expf(-pow((x - mu), 2) / powf(sigma, 2));

			// renormalize to <0, 1>
			//outputPtr[j] = fminf(fmaxf(t, 0), 1);
			outputPtr[j] = sigmoid(mu + x * sigma);
		}
	}
}
