#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

extern "C"
{

	__device__ int indexFromXY(int x, int y, int width)
	{
		return y * width + x;
	}

	__global__ void ConvolutionForwardKernel(
		float *inputPtr,
		float *filterPtr,
		float *biasPtr,
		float *outputPtr,
		int filterWidth, int filterHeight,
		int filterDepth,
		int filterSliceSize, // one layer of filter volume, fW * fH
		int filterSize, // one filter volume, fW * fH * inputDepth
		int inputSliceSize, // one layer of input data, e.g. one channel of an RGB image
		int inputWidth,
		int outputSize, // size of one resulting output layer = one learned filter, oW * oH (there are filterCount of these)
		int filtersPerRow,
		int horStride, int verStride,
		int thisLayerSize
		)
	{
		int idx = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (idx < thisLayerSize)
		{
			int filterIdx = idx / outputSize;
			
			int inputTileX = (idx % outputSize) % filtersPerRow;
			int inputTileY = (idx % outputSize) / filtersPerRow;

			
			float result = 0;

			for (size_t z = 0; z < filterDepth; z++) // Z
			{
				int inputIndex = z * inputSliceSize;
				int y = inputTileY * verStride;

				for (size_t j = 0; j < filterHeight; j++) // Y
				{
					int x = inputTileX * horStride;
					int filterIndex = filterSize * filterIdx + z * filterSliceSize;

					for (size_t i = 0; i < filterWidth; i++) // X
					{
						result +=
							inputPtr[inputIndex + indexFromXY(x, y, inputWidth)] * // input
							filterPtr[filterIndex + indexFromXY(i, j, filterWidth)]; // weight
						++x;
					}
					++y;

				}
			}

			result += biasPtr[filterIdx];

			outputPtr[idx] = result;

		}
	}


	// computes deltas
	__global__ void ConvolutionBackwardKernel(
		float *filterPtr,
		float *deltaPtr,
		float *nextLayerDeltaPtr,
		int filterWidth, int filterHeight,
		int filterDepth,
		int filterSliceSize, // one layer of filter volume, fW * fH
		int filterSize, // one filter volume, fW * fH * inputDepth
		int inputSliceSize, // one layer of input data, e.g. one channel of an RGB image
		int inputWidth,
		int outputSize, // size of one resulting output layer = one learned filter, oW * oH (there are filterCount of these)
		int filtersPerRow,
		int horStride, int verStride,
		int thisLayerSize
		)
	{
		int idx = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (idx < thisLayerSize)
		{
			int filterIdx = idx / outputSize;

			int inputTileX = (idx % outputSize) % filtersPerRow;
			int inputTileY = (idx % outputSize) / filtersPerRow;


			float result = 0;

			for (size_t z = 0; z < filterDepth; z++) // Z
			{
				int y = inputTileY * verStride;
				for (size_t j = 0; j < filterHeight; j++) // Y
				{
					int x = inputTileX * horStride;
					for (size_t i = 0; i < filterWidth; i++) // X
					{
						result +=
							filterPtr[filterSize * filterIdx + z * filterSliceSize + indexFromXY(filterWidth - 1 - i, filterHeight - 1 - j, filterWidth)] *
							nextLayerDeltaPtr[z * inputSliceSize + indexFromXY(x, y, inputWidth)];
						++x;
					}
					++y;
				}
			}

			deltaPtr[idx] = result;

		}
	}

	__global__ void PadImageKernel(
		float *inputPtr,
		float *outputPtr,
		int inputWidth,
		int pad,
		int inputSize, // one depth slice / one layer / one color channel
		int outputSize,
		int totalInputSize // whole image (all color channels combined)
		)
	{
		int idx = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (idx < totalInputSize)
		{
			int depth = idx / inputSize;

			int outputDepthShift = depth * outputSize;

			int rowIdx = (idx % inputSize) / inputWidth;
			int colIdx = (idx % inputSize) % inputWidth;

			outputPtr[indexFromXY(pad + colIdx, pad + rowIdx, pad + inputWidth + pad) + outputDepthShift] = inputPtr[idx];
		}
	}
}