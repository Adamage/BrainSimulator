#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

extern "C"
{

	__device__ int indexFromXY (int x, int y, int width)
	{
		return y * width + x;
	}

	__global__ void PoolingForwardKernel (
		float *inputPtr,
		float *outputPtr,
		int *activatedNeuronsPtr,
		int inputWidth, int inputSize,
		int filterWidth, int filterHeight,
		int horStride, int verStride,
		int outputWidth, int outputSize,
		int thisLayerSize
	)
	{
		int idx = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
				+ blockDim.x * blockIdx.x				//blocks preceeding current block
				+ threadIdx.x;

		if (idx < thisLayerSize)
		{

			int depth = idx / outputSize;
			int depthShift = depth * inputSize;


			int inputTileX = (idx % outputSize) % outputWidth;
			int inputTileY = (idx % outputSize) / outputWidth;
			

			int y = inputTileY * verStride;
			int maxY = y;

			int maxX = inputTileX * horStride;

			
			float maxValue = inputPtr[depthShift + indexFromXY(maxX, y, inputWidth)];

			for (int j = 0; j < filterHeight; j++)
			{
				int x = inputTileX * horStride;
				for (int i = 0; i < filterWidth; i++)
				{
					float value = inputPtr[depthShift + indexFromXY(x, y, inputWidth)];
					if (value > maxValue) {
						value = maxValue;
						maxX = x;
						maxY = y;
					}
					++x;
				}
				++y;
			}

			outputPtr[idx] = maxValue;
			activatedNeuronsPtr[idx] = depthShift + indexFromXY(maxX, maxY, inputWidth);
		}
	}


	__global__ void PoolingBackwardKernel (
		float *thisLayerDelta,
		float *prevLayerDelta,
		int *activatedNeuronsPtr,
		int thisLayerSize
	)
	{
		int idx = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
				+ blockDim.x * blockIdx.x				//blocks preceeding current block
				+ threadIdx.x;

		if (idx < thisLayerSize)
		{
			prevLayerDelta[activatedNeuronsPtr[idx]] = thisLayerDelta[idx];
		}
	}
}