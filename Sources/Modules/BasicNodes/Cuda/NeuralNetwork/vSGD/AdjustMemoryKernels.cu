#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

extern "C"
{
	__global__ void FullyConnectedAdjustMemoryKernel(
		float *weightsGradPtr,
		float *biasGradPtr,
		float *weightGradCurvePtr,
		float *biasGradCurvePtr,
		float *avgWeightGradPtr,
		float *avgBiasGradPtr,
		float *avgWeightGradVarPtr,
		float *avgBiasGradVarPtr,
		float *avgWeightGradCurvePtr,
		float *avgBiasGradCurvePtr,
		float *avgWeightGradCurveVarPtr,
		float *avgBiasGradCurveVarPtr,
		float *weightMemorySizePtr,
		float *biasMemorySizePtr,
		float *dropoutMaskPtr,
		int prevLayerSize,
		int thisLayerSize
		)
	{
		// i: prev. layer neuron id
		// j: current layer neuron id
		int i;
		int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (j < thisLayerSize)
		{
			if (!dropoutMaskPtr[j])
			{
				int index = j;
				for (i = 0; i < prevLayerSize; i++)
				{
					// check for weight outliers
					if (
						abs(weightsGradPtr[index] - avgWeightGradPtr[index]) > 2 * sqrtf(avgWeightGradVarPtr[index]) - avgWeightGradPtr[index] * avgWeightGradPtr[index] ||
						abs(weightGradCurvePtr[index] - avgWeightGradCurvePtr[index]) > 2 * sqrtf(avgWeightGradCurveVarPtr[index] - avgWeightGradCurveVarPtr[index] * avgWeightGradCurveVarPtr[index])
						)
						// TODO: test which one works best
						//weightMemorySizePtr[index] += 1; // original method suggested in http://arxiv.org/pdf/1301.3764.pdf
						weightMemorySizePtr[index] = 2.2f; // reset to 2.2 according to the Adasecant method in http://arxiv.org/pdf/1412.7419v4.pdf

					index += thisLayerSize;
				}

				// check for bias outliers
				if (
					abs(biasGradPtr[j] - avgBiasGradPtr[j]) > 2 * sqrtf(avgBiasGradVarPtr[j]) - avgBiasGradPtr[j] * avgBiasGradPtr[j] ||
					abs(biasGradCurvePtr[j] - avgBiasGradCurvePtr[j]) > 2 * sqrtf(avgBiasGradCurveVarPtr[j] - avgBiasGradCurveVarPtr[j] * avgBiasGradCurveVarPtr[j])
					)
					// TODO: test which one works best
					//biasMemorySizePtr[j] += 1; // original method suggested in http://arxiv.org/pdf/1301.3764.pdf
					biasMemorySizePtr[j] = 2.2f; // reset to 2.2 according to the Adasecant method in http://arxiv.org/pdf/1412.7419v4.pdf
			}
		}
	}

	__global__ void FullyConnectedUpdateMemoryKernel(
		float *avgWeightGradPtr,
		float *avgBiasGradPtr,
		float *avgWeightGradVarPtr,
		float *avgBiasGradVarPtr,
		float *weightMemorySizePtr,
		float *biasMemorySizePtr,
		float *dropoutMaskPtr,
		int prevLayerSize,
		int thisLayerSize
		)
	{
		// i: prev. layer neuron id
		// j: current layer neuron id
		int i;
		int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (j < thisLayerSize)
		{
			if (!dropoutMaskPtr[j])
			{
				int index = j;
				for (i = 0; i < prevLayerSize; i++)
				{
					// update memory size
					weightMemorySizePtr[index] = (1.0f - avgWeightGradPtr[index] * avgWeightGradPtr[index] / avgWeightGradVarPtr[index]) * weightMemorySizePtr[index] + 1.0f;

					index += thisLayerSize;
				}

				// update memory size
				biasMemorySizePtr[j] = (1.0f - avgBiasGradPtr[j] * avgBiasGradPtr[j] / avgBiasGradVarPtr[j]) * biasMemorySizePtr[j] + 1.0f;
			}
		}
	}
}