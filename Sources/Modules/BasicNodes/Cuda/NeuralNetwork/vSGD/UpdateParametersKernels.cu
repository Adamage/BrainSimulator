#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

extern "C"
{
	__global__ void FullyConnectedUpdateParametersKernel(
		float *weightPtr,
		float *biasPtr,
		float *weightLearningRatePtr,
		float *biasLearningRatePtr,
		float *weightGradientPtr,
		float *biasGradientPtr,
		float *dropoutMaskPtr,
		int prevLayerSize,
		int thisLayerSize
		)
	{
		// i: prev. layer neuron id
		// j: current layer neuron id
		int i;
		int j = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (j < thisLayerSize)
		{
			if (!dropoutMaskPtr[j])
			{
				// update weights
				int index = j;
				for (i = 0; i < prevLayerSize; i++)
				{
					weightPtr[index] -= weightLearningRatePtr[index] * weightGradientPtr[index];

					index += thisLayerSize;
				}

				// update bias
				biasPtr[j] -= biasLearningRatePtr[j] * biasGradientPtr[j];
			}
		}
	}
}