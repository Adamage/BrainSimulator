#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

#include "../NeuralNetwork/Activation/ActivationFunction.cu"



extern "C"
{
	__global__ void LSTMDeltaKernel(
		float *cellStateErrors,
		float *outputGateDeltas,
		float *cellStates,
		float *outputGateActivations,
		float *outputGateActivationDerivatives,
		float *nextLayerDeltas,
		float *nextLayerWeights,

		int nextLayerSize,
		int cellCount,
		int cellsPerBlock
		)
	{
		int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (memoryBlockId < cellCount / cellsPerBlock)
		{
			float outputGateDeltaSum = 0.0;

			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				float sum = 0.0;
				int index = memoryBlockId * nextLayerSize;
				for (int j = 0; j < nextLayerSize; j++)
				{
					sum += nextLayerWeights[index + j] * nextLayerDeltas[j];
				}
				cellStateErrors[cellId] = outputGateActivations[memoryBlockId] * sum;
				outputGateDeltaSum += cellStates[cellId] * sum;
			}

			outputGateDeltas[memoryBlockId] = outputGateActivationDerivatives[memoryBlockId] * outputGateDeltaSum;
		}
	}
}
