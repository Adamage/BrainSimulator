#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

#include "../NeuralNetwork/Activation/ActivationFunction.cu"



extern "C"
{
	__global__ void LSTMDeltaKernel(
		float *cellStateErrors,
		float *outputGateDeltas,
		float *cellStates,
		float *outputGateActivations,
		float *outputGateActivationDerivatives,
		float *deltas,

		int cellCount,
		int cellsPerBlock
		)
	{
		int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (memoryBlockId < cellCount / cellsPerBlock)
		{
			float outputGateDeltaSum = 0.0;

			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				float delta = -deltas[cellId];
				cellStateErrors[cellId] = outputGateActivations[memoryBlockId] * delta;
				outputGateDeltaSum += cellStates[cellId] * delta;
			}

			outputGateDeltas[memoryBlockId] = outputGateActivationDerivatives[memoryBlockId] * outputGateDeltaSum;
		}
	}

	__global__ void LSTMDeltaBackKernel(
		ActivationFunctionEnum prevLayerActivationFunction,
		float *prevWeighedInputPtr,
		float *prevDeltaPtr,
		float *cellStateErrors,
		float *inputGateActivations,
		float *cellInputWeights,

		int prevLayerNeurons,
		int cellCount,
		int cellsPerBlock
		)
	{
		int neuronId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (neuronId < prevLayerNeurons)
		{
			float deltaSum = 0.0f;

			for (int memoryBlockId = 0; memoryBlockId < cellCount / cellsPerBlock; memoryBlockId++)
			{
				float inputGate = inputGateActivations[memoryBlockId];
				for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
				{
					deltaSum += cellInputWeights[cellId * (prevLayerNeurons + cellCount + 1) + neuronId] * inputGate * cellStateErrors[cellId];
				}
			}

			prevDeltaPtr[neuronId] = -deltaSum * EvaluateDerivative(prevLayerActivationFunction, prevWeighedInputPtr[neuronId]);
		}
	}
}
