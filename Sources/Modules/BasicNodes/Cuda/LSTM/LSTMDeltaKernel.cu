#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

#include "../NeuralNetwork/Activation/ActivationFunction.cu"



extern "C"
{
	__global__ void LSTMDeltaKernelBPTT(
		float* deltas,
		float* cellStates,
		float* previousCellStates,
		float* cellStateErrors,

		float* outputGateDeltas,
		float* forgetGateDeltas,
		float* inputGateDeltas,
        float* cellInputDeltas,

		float* outputGateActivations,
		float* forgetGateActivations,
		float* inputGateActivations,

		float* cellInputActivationDerivatives,
		float* outputGateActivationDerivatives,
		float* forgetGateActivationDerivatives,
		float* inputGateActivationDerivatives,

		float* cellInputWeights,
		float* outputGateWeights,
		float* forgetGateWeights,
		float* inputGateWeights,

		int inputCount,
		int cellCount,
		int cellsPerBlock
		)
	{
		int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (memoryBlockId < cellCount / cellsPerBlock)
		{
			float outputGateDeltaSum = 0.0;

			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				outputGateDeltaSum += cellStates[cellId] * -deltas[cellId];
			}
			outputGateDeltas[memoryBlockId] = outputGateActivationDerivatives[memoryBlockId] * outputGateDeltaSum;

			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				cellStateErrors[cellId] = -deltas[cellId] * outputGateActivations[memoryBlockId] * cellStates[cellId] +
					cellStateErrors[cellId] * forgetGateActivations[cellId] +
					inputGateDeltas[cellId] * inputGateWeights[(memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1)) + inputCount + cellCount] +
					forgetGateDeltas[cellId] * forgetGateWeights[(memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1)) + inputCount + cellCount] +
					outputGateDeltas[cellId] * outputGateWeights[(memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1)) + inputCount + cellCount];

                cellInputDeltas[cellId] = inputGateActivations[memoryBlockId] *  cellInputActivationDerivatives[memoryBlockId] * cellStateErrors[cellId];
            }

			inputGateDeltas[memoryBlockId] = 0;
			forgetGateDeltas[memoryBlockId] = 0;

			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				inputGateDeltas[memoryBlockId] += inputGateActivationDerivatives[memoryBlockId] * cellStateErrors[cellId] * inputGateActivations[memoryBlockId];
				forgetGateDeltas[memoryBlockId] += forgetGateActivationDerivatives[memoryBlockId] * cellStateErrors[cellId] * previousCellStates[cellId];
			}
		}
	}

	__device__ float GateDeltaBack(float* prevDeltaPtr, float* gateDeltas, float* gateWeights,int neuronId, int cellCountDevcellsPerBlock) // ???? IS TI CORRECT????
	{
		for (int memoryBlockId = 0; memoryBlockId < cellCountDevcellsPerBlock; memoryBlockId++)
		{
			prevDeltaPtr[neuronId] += -gateDeltas[memoryBlockId] * gateWeights[neuronId];
		}
	}

	__global__ void LSTMDeltaBackKernelBPPT(
		ActivationFunctionEnum prevLayerActivationFunction,
		float *prevDeltaPtr,

		float* outputGateDeltas,
		float* forgetGateDeltas,
		float* inputGateDeltas,

		float *cellInputWeights,
		float *inputGateWeights,
		float *forgetGateWeights,
		float *outputGateWeights,

		int prevLayerNeurons,
		int cellCount,
		int cellsPerBlock
		)
	{
		int neuronId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (neuronId < prevLayerNeurons)
		{
			GateDeltaBack(prevDeltaPtr, inputGateDeltas, inputGateWeights,neuronId, cellCount / cellsPerBlock);
			GateDeltaBack(prevDeltaPtr, forgetGateDeltas, forgetGateWeights,neuronId, cellCount / cellsPerBlock);
			GateDeltaBack(prevDeltaPtr, outputGateDeltas, outputGateWeights,neuronId, cellCount / cellsPerBlock);
		}
	}



	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
    /*
    /*  ORIGINAL FROM KAREL
     */
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/



	__global__ void LSTMDeltaKernel(
		float *cellStateErrors,
		float *outputGateDeltas,
		float *cellStates,
		float *outputGateActivations,
		float *outputGateActivationDerivatives,
		float *deltas,

		int cellCount,
		int cellsPerBlock
		)
	{
		int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (memoryBlockId < cellCount / cellsPerBlock)
		{
			float outputGateDeltaSum = 0.0;

			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				float delta = -deltas[cellId];
				cellStateErrors[cellId] = outputGateActivations[memoryBlockId] * delta;
				outputGateDeltaSum += cellStates[cellId] * delta;
			}

			outputGateDeltas[memoryBlockId] = outputGateActivationDerivatives[memoryBlockId] * outputGateDeltaSum;
		}
	}

	__global__ void LSTMDeltaBackKernel(
		ActivationFunctionEnum prevLayerActivationFunction,
		float *prevWeighedInputPtr,
		float *prevDeltaPtr,
		float *cellStateErrors,
		float *previousCellStates,
		float *inputGateActivations,

		float *cellInputActivationDerivatives,
		float *inputGateActivationDerivatives,
		float *forgetGateActivationDerivatives,

		float *cellInputWeights,
		float *inputGateWeights,
		float *forgetGateWeights,
		float *outputGateWeights,

		float *outputGateDeltas,

		int prevLayerNeurons,
		int cellCount,
		int cellsPerBlock
		)
	{
		int neuronId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (neuronId < prevLayerNeurons)
		{
			float delta = 0.0f;

			for (int memoryBlockId = 0; memoryBlockId < cellCount / cellsPerBlock; memoryBlockId++)
			{
				float inputGateError = 0.0f;
				float forgetGateError = 0.0f;

				for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
				{
					inputGateError += inputGateActivationDerivatives[memoryBlockId] * cellStateErrors[cellId] * inputGateActivations[memoryBlockId];
					forgetGateError += forgetGateActivationDerivatives[memoryBlockId] * cellStateErrors[cellId] * previousCellStates[cellId];
					// cell input error
					delta += cellInputWeights[cellId * (prevLayerNeurons + cellCount + 1) + neuronId] * inputGateActivations[memoryBlockId] * cellStateErrors[cellId] * cellInputActivationDerivatives[cellId];
				}

				delta += inputGateWeights[memoryBlockId * (prevLayerNeurons + cellCount + cellsPerBlock + 1) + neuronId] * inputGateError;
				delta += forgetGateWeights[memoryBlockId * (prevLayerNeurons + cellCount + cellsPerBlock + 1) + neuronId] * forgetGateError;
				delta += outputGateWeights[memoryBlockId * (prevLayerNeurons + cellCount + cellsPerBlock + 1) + neuronId] * outputGateDeltas[memoryBlockId];
			}

			prevDeltaPtr[neuronId] = -delta * EvaluateDerivative(prevLayerActivationFunction, prevWeighedInputPtr[neuronId]);
		}
	}
 
  
 

}
