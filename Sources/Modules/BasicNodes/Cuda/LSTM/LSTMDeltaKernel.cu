#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

#include "../NeuralNetwork/Activation/ActivationFunction.cu"



extern "C"
{
	__global__ void LSTMDeltaKernelBPTT(
		float* deltas,
		float* cellStates,
		float* previousCellStates,
		float* cellStateErrors,

		float* outputGateDeltas,
		float* forgetGateDeltas,
		float* inputGateDeltas,
        float* cellInputDeltas,

		float* outputGateActivations,
		float* forgetGateActivations,
		float* inputGateActivations,

		float* cellInputActivationDerivatives,
		float* outputGateActivationDerivatives,
		float* forgetGateActivationDerivatives,
		float* inputGateActivationDerivatives,

		float* cellInputWeights,
		float* outputGateWeights,
		float* forgetGateWeights,
		float* inputGateWeights,

		int inputCount,
		int cellCount,
		int cellsPerBlock
		)
	{
		int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (memoryBlockId < cellCount / cellsPerBlock)
		{
			float outputGateDeltaSum = 0.0;

			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				outputGateDeltaSum += cellStates[cellId] * -deltas[cellId];
			}
			outputGateDeltas[memoryBlockId] = outputGateActivationDerivatives[memoryBlockId] * outputGateDeltaSum;

			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				int peepHoleWeightId = (memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1)) + inputCount + cellCount + cellId;
				cellStateErrors[cellId] = -deltas[cellId] * outputGateActivations[memoryBlockId] * cellStates[cellId] +
					cellStateErrors[cellId] * forgetGateActivations[cellId] +
					inputGateDeltas[memoryBlockId] * inputGateWeights[peepHoleWeightId] +
					forgetGateDeltas[memoryBlockId] * forgetGateWeights[peepHoleWeightId] +
					outputGateDeltas[memoryBlockId] * outputGateWeights[peepHoleWeightId];

                cellInputDeltas[cellId] = inputGateActivations[memoryBlockId] *  cellInputActivationDerivatives[memoryBlockId] * cellStateErrors[cellId];
            }

			inputGateDeltas[memoryBlockId] = 0;
			forgetGateDeltas[memoryBlockId] = 0;

			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				inputGateDeltas[memoryBlockId] += inputGateActivationDerivatives[memoryBlockId] * cellStateErrors[cellId] * inputGateActivations[memoryBlockId];
				forgetGateDeltas[memoryBlockId] += forgetGateActivationDerivatives[memoryBlockId] * cellStateErrors[cellId] * previousCellStates[cellId];
			}
		}
	}

	__global__ void LSTMGateGradientKernelBPTT(
		float *input,
		float *previousOutput,
		float *cellStates,

		float *inputGateDeltas,
		float *forgetGateDeltas,
		float *outputGateDeltas,

		float* outputGateWeightGradient,
		float* inputGateWeightGradient,
		float* forgetGateWeightGradient,

		int inputCount,
		int previousOutputCount,
		int cellsPerBlock
		)
	{
		int weightId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		int weightsPerGate = inputCount + previousOutputCount + cellsPerBlock + 1;

		if (weightId < weightsPerGate * previousOutputCount / cellsPerBlock)
		{
			int fromId = weightId % weightsPerGate;
			int toId = weightId / weightsPerGate;

			//calculate output gate weight gradient
			int isFromInputUnit = fromId >= 0 && fromId < inputCount;
			int isFromPreviousOutputUnit = (fromId >= inputCount) && (fromId < inputCount + previousOutputCount);
			int isPeephole = (fromId >= inputCount + previousOutputCount) && (fromId < inputCount + previousOutputCount + cellsPerBlock);
			int isFromBiasUnit = fromId == (inputCount + previousOutputCount + cellsPerBlock);

			float inputFromWeight = isFromInputUnit * input[isFromInputUnit * fromId]
				+ isFromPreviousOutputUnit * previousOutput[isFromPreviousOutputUnit * (fromId - inputCount)]
				+ isPeephole * cellStates[isPeephole * (toId * cellsPerBlock + (fromId - inputCount - previousOutputCount))]
				+ isFromBiasUnit * 1;

			outputGateWeightGradient[weightId] = outputGateDeltas[toId] * inputFromWeight;
			inputGateWeightGradient[weightId] = inputGateDeltas[toId] * inputFromWeight;
			forgetGateWeightGradient[weightId] = forgetGateDeltas[toId] * inputFromWeight;
		}
	}

	__global__ void LSTMCellInputGradientKernelBPTT(
		float *input,
		float *previousOutput,

		float *cellInputDeltas,
		float *cellInputWeightGradient,

		int inputCount,
		int previousOutputCount,
		int cellsPerBlock
		)
	{
		int weightId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		int weightsPerCell = inputCount + previousOutputCount + 1;

		if (weightId < weightsPerCell * previousOutputCount)
		{
			int fromId = weightId % weightsPerCell;
			int toId = weightId / weightsPerCell;

			int isFromInputUnit = fromId >= 0 && fromId < inputCount;
			int isFromPreviousOutputUnit = (fromId >= inputCount) && (fromId < inputCount + previousOutputCount);
			int isFromBiasUnit = fromId == (inputCount + previousOutputCount);

			float inputFromWeight = isFromInputUnit * input[isFromInputUnit * fromId]
				+ isFromPreviousOutputUnit * previousOutput[isFromPreviousOutputUnit * (fromId - inputCount)]
				+ isFromBiasUnit * 1;

			cellInputWeightGradient[weightId] = cellInputDeltas[toId] * inputFromWeight;
		}
	}

	__global__ void LSTMDeltaBackKernelBPTT(
		ActivationFunctionEnum prevLayerActivationFunction,
		float *prevDeltaPtr,

		float* cellInputDeltas,
		float* outputGateDeltas,
		float* forgetGateDeltas,
		float* inputGateDeltas,

		float *cellInputWeights,
		float *inputGateWeights,
		float *forgetGateWeights,
		float *outputGateWeights,

		int prevLayerNeurons,
		int cellCount,
		int cellsPerBlock
		)
	{
		int neuronId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		int weightsPerCell = prevLayerNeurons + cellCount + 1;
		int weightsPerGate = prevLayerNeurons + cellCount + cellsPerBlock + 1;

		if (neuronId < prevLayerNeurons)
		{
			int memoryBlockCount = cellCount / cellsPerBlock;
			for (int memoryBlockId = 0; memoryBlockId < memoryBlockCount; memoryBlockId++)
			{
				int cellWeightId = memoryBlockId * weightsPerCell + neuronId;
				int gateWeightId = memoryBlockId * weightsPerGate + neuronId;

				prevDeltaPtr[neuronId] += -cellInputDeltas[memoryBlockId] * cellInputWeights[cellWeightId];
				prevDeltaPtr[neuronId] += -inputGateDeltas[memoryBlockId] * inputGateWeights[gateWeightId];
				prevDeltaPtr[neuronId] += -forgetGateDeltas[memoryBlockId] * forgetGateWeights[gateWeightId];
				prevDeltaPtr[neuronId] += -outputGateDeltas[memoryBlockId] * outputGateWeights[gateWeightId];
			}
		}
	}



	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
    /*
    /*  ORIGINAL FROM KAREL
     */
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/



	__global__ void LSTMDeltaKernel(
		float *cellStateErrors,
		float *outputGateDeltas,
		float *cellStates,
		float *outputGateActivations,
		float *outputGateActivationDerivatives,
		float *deltas,

		int cellCount,
		int cellsPerBlock
		)
	{
		int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (memoryBlockId < cellCount / cellsPerBlock)
		{
			float outputGateDeltaSum = 0.0;

			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				float delta = -deltas[cellId];
				cellStateErrors[cellId] = outputGateActivations[memoryBlockId] * delta;
				outputGateDeltaSum += cellStates[cellId] * delta;
			}

			outputGateDeltas[memoryBlockId] = outputGateActivationDerivatives[memoryBlockId] * outputGateDeltaSum;
		}
	}

	__global__ void LSTMDeltaBackKernel(
		ActivationFunctionEnum prevLayerActivationFunction,
		float *prevWeighedInputPtr,
		float *prevDeltaPtr,
		float *cellStateErrors,
		float *previousCellStates,
		float *inputGateActivations,

		float *cellInputActivationDerivatives,
		float *inputGateActivationDerivatives,
		float *forgetGateActivationDerivatives,

		float *cellInputWeights,
		float *inputGateWeights,
		float *forgetGateWeights,
		float *outputGateWeights,

		float *outputGateDeltas,

		int prevLayerNeurons,
		int cellCount,
		int cellsPerBlock
		)
	{
		int neuronId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (neuronId < prevLayerNeurons)
		{
			float delta = 0.0f;

			for (int memoryBlockId = 0; memoryBlockId < cellCount / cellsPerBlock; memoryBlockId++)
			{
				float inputGateError = 0.0f;
				float forgetGateError = 0.0f;

				for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
				{
					inputGateError += inputGateActivationDerivatives[memoryBlockId] * cellStateErrors[cellId] * inputGateActivations[memoryBlockId];
					forgetGateError += forgetGateActivationDerivatives[memoryBlockId] * cellStateErrors[cellId] * previousCellStates[cellId];
					// cell input error
					delta += cellInputWeights[cellId * (prevLayerNeurons + cellCount + 1) + neuronId] * inputGateActivations[memoryBlockId] * cellStateErrors[cellId] * cellInputActivationDerivatives[cellId];
				}

				delta += inputGateWeights[memoryBlockId * (prevLayerNeurons + cellCount + cellsPerBlock + 1) + neuronId] * inputGateError;
				delta += forgetGateWeights[memoryBlockId * (prevLayerNeurons + cellCount + cellsPerBlock + 1) + neuronId] * forgetGateError;
				delta += outputGateWeights[memoryBlockId * (prevLayerNeurons + cellCount + cellsPerBlock + 1) + neuronId] * outputGateDeltas[memoryBlockId];
			}

			prevDeltaPtr[neuronId] = -delta * EvaluateDerivative(prevLayerActivationFunction, prevWeighedInputPtr[neuronId]);
		}
	}
 
  
 

}
