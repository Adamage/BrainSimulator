#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

#include "../NeuralNetwork/Activation/ActivationFunction.cu"



extern "C"
{
	__device__ float Clip(float value, float clip)
	{
		return (clip == 0) * value + (clip != 0) * ((value > clip) * clip + (value < -clip) * -clip + (value >= -clip && value <= clip) * value);

		/* avoids thread divergence, equivalent to:
		if (clip == 0)
			return value;
		else if (value > clip)
			return clip;
		else if (value < -clip)
			return -clip;
		else
			return value;
		*/
	}

	__device__ float GetNetInput(
		int memoryBlockId,
		int cellsPerBlock,
		float* weights,
		int weightsOffset,
		float *input,
		int inputCount,
		float *previousOutput,
		int previousOutputCount,
		float *cellStates,
		bool peephole,
		bool bias
		)
	{
		int weightId = weightsOffset;
		float netInput = 0;

		// signal from external input
		for (int i = 0; i < inputCount; i++)
		{
			netInput += weights[weightId] * input[i];
			weightId++;
		}

		// signal from previous output of memory blocks
		for (int i = 0; i < previousOutputCount; i++)
		{
			netInput += weights[weightId] * previousOutput[i];
			weightId++;
		}

		// signal from peephole connections
		if (peephole)
		{
			for (int i = 0; i < cellsPerBlock; i++)
			{
				netInput += weights[weightId] * cellStates[memoryBlockId * cellsPerBlock + i];
				weightId++;
			}
		}

		if (bias)
		{
			netInput += weights[weightId];
		}

		return netInput;
	}


	__global__ void LSTMFeedForwardKernelBPTT(
		ActivationFunctionEnum inputActivationFunction,
		ActivationFunctionEnum gateActivationFunction,
		float *input,
		float *output,
		float *previousOutput,
		float *cellStates,
		float *cellStatesActivations,
		float *cellStateActivationDerivatives,
		float *previousCellStates,
		float *cellInputActivations,
		float *cellInputActivationDerivatives,
		float *inputGateActivations,
		float *inputGateActivationDerivatives,
		float *forgetGateActivations,
		float *forgetGateActivationDerivatives,
		float *outputGateActivations,
		float *outputGateActivationDerivatives,

		float *cellInputWeights,
		float *inputGateWeights,
		float *forgetGateWeights,
		float *outputGateWeights,

		float clipCellState,

		int inputCount,
		int cellCount,
		int cellsPerBlock
		)
	{
		int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (memoryBlockId < cellCount / cellsPerBlock)
		{

			// step 1: calculate activations of input and forget gate
			float inputGateNetInput = GetNetInput(
				memoryBlockId,
				cellsPerBlock,
				inputGateWeights,
				memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1),
				input,
				inputCount,
				previousOutput,
				cellCount,
				previousCellStates,
				true,
				true
			);
			float forgetGateNetInput = GetNetInput(
				memoryBlockId,
				cellsPerBlock,
				forgetGateWeights,
				memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1),
				input,
				inputCount,
				previousOutput,
				cellCount,
				previousCellStates,
				true,
				true
			);

			// activation function of all gates must be in range [0,1], sigmoid activation function is used
			float inputGateActivation = Evaluate(gateActivationFunction, inputGateNetInput);
			float forgetGateActivation = Evaluate(gateActivationFunction, forgetGateNetInput);

			inputGateActivations[memoryBlockId] = inputGateActivation;
			forgetGateActivations[memoryBlockId] = forgetGateActivation;

			inputGateActivationDerivatives[memoryBlockId] = EvaluateDerivative(gateActivationFunction, inputGateNetInput);
			forgetGateActivationDerivatives[memoryBlockId] = EvaluateDerivative(gateActivationFunction, forgetGateNetInput);

			// step 2: calculate activation of memory block's cells
			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				float cellNetInput = GetNetInput(
					memoryBlockId,
					cellsPerBlock,
					cellInputWeights,
					cellId * (inputCount + cellCount + 1),
					input,
					inputCount,
					previousOutput,
					cellCount,
					NULL,
					false,
					true
				);

				float cellInputActivation = Evaluate(inputActivationFunction, cellNetInput);

				cellInputActivations[cellId] = cellInputActivation;
				cellInputActivationDerivatives[cellId] = EvaluateDerivative(inputActivationFunction, cellNetInput);

				cellStates[cellId] = Clip(forgetGateActivation * previousCellStates[cellId] + inputGateActivation * cellInputActivation, clipCellState);
			}

			// step 3: calculate output gate activation
			float outputGateNetInput = GetNetInput(
				memoryBlockId,
				cellsPerBlock,
				outputGateWeights,
				memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1),
				input,
				inputCount,
				previousOutput,
				cellCount,
				cellStates,
				true,
				true
			);

			float outputGateActivation = Evaluate(gateActivationFunction, outputGateNetInput);
			outputGateActivations[memoryBlockId] = outputGateActivation;
			outputGateActivationDerivatives[memoryBlockId] = EvaluateDerivative(gateActivationFunction, outputGateNetInput);

			// step 4: calculate output of all memory block's cells
			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				cellStatesActivations[cellId] = Evaluate(inputActivationFunction, cellStates[cellId]);
				cellStateActivationDerivatives[cellId] = EvaluateDerivative(inputActivationFunction, cellStates[cellId]);

				output[cellId] = outputGateActivation * cellStatesActivations[cellId];
			}
		}
	}




	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
    /*
    /*  ORIGINAL FROM KAREL
     */
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/
	/*****************************************************************************************************************************************************************/



	__global__ void LSTMFeedForwardKernel(
		ActivationFunctionEnum inputActivationFunction,
		ActivationFunctionEnum gateActivationFunction,
		float *input,
		float *output,
		float *previousOutput,
		float *cellStates,
		float *previousCellStates,
		float *cellInputActivations,
		float *cellInputActivationDerivatives,
		float *inputGateActivations,
		float *inputGateActivationDerivatives,
		float *forgetGateActivations,
		float *forgetGateActivationDerivatives,
		float *outputGateActivations,
		float *outputGateActivationDerivatives,

		float *cellInputWeights,
		float *inputGateWeights,
		float *forgetGateWeights,
		float *outputGateWeights,

		float clipCellState,

		int inputCount,
		int cellCount,
		int cellsPerBlock
		)
	{
		int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x	//rows preceeding current row in grid
			+ blockDim.x * blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (memoryBlockId < cellCount / cellsPerBlock)
		{

			// step 1: calculate activations of input and forget gate
			float inputGateNetInput = GetNetInput(
				memoryBlockId,
				cellsPerBlock,
				inputGateWeights,
				memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1),
				input,
				inputCount,
				previousOutput,
				cellCount,
				previousCellStates,
				true,
				true
			);
			float forgetGateNetInput = GetNetInput(
				memoryBlockId,
				cellsPerBlock,
				forgetGateWeights,
				memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1),
				input,
				inputCount,
				previousOutput,
				cellCount,
				previousCellStates,
				true,
				true
			);

			// activation function of all gates must be in range [0,1], sigmoid activation function is used
			float inputGateActivation = Evaluate(gateActivationFunction, inputGateNetInput);
			float forgetGateActivation = Evaluate(gateActivationFunction, forgetGateNetInput);

			inputGateActivations[memoryBlockId] = inputGateActivation;
			forgetGateActivations[memoryBlockId] = forgetGateActivation;

			inputGateActivationDerivatives[memoryBlockId] = EvaluateDerivative(gateActivationFunction, inputGateNetInput);
			forgetGateActivationDerivatives[memoryBlockId] = EvaluateDerivative(gateActivationFunction, forgetGateNetInput);

			// step 2: calculate activation of memory block's cells
			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				float cellNetInput = GetNetInput(
					memoryBlockId,
					cellsPerBlock,
					cellInputWeights,
					cellId * (inputCount + cellCount + 1),
					input,
					inputCount,
					previousOutput,
					cellCount,
					NULL,
					false,
					true
				);

				float cellInputActivation = Evaluate(inputActivationFunction, cellNetInput);

				cellInputActivations[cellId] = cellInputActivation;
				cellInputActivationDerivatives[cellId] = EvaluateDerivative(inputActivationFunction, cellNetInput);

				cellStates[cellId] = Clip(forgetGateActivation * previousCellStates[cellId] + inputGateActivation * cellInputActivation, clipCellState);
			}

			// step 3: calculate output gate activation
			float outputGateNetInput = GetNetInput(
				memoryBlockId,
				cellsPerBlock,
				outputGateWeights,
				memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1),
				input,
				inputCount,
				previousOutput,
				cellCount,
				cellStates,
				true,
				true
			);

			float outputGateActivation = Evaluate(gateActivationFunction, outputGateNetInput);
			outputGateActivations[memoryBlockId] = outputGateActivation;
			outputGateActivationDerivatives[memoryBlockId] = EvaluateDerivative(gateActivationFunction, outputGateNetInput);

			// step 4: calculate output of all memory block's cells
			for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
			{
				output[cellId] = outputGateActivation * cellStates[cellId];
			}
		}
	}
}
