#include "hip/hip_runtime.h"
#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

#include <hip/hip_runtime.h> 
#include <> 
#include <> 
#include <builtin_types.h> 
#include <vector_functions.h> 
#include <float.h>


extern "C"  
{
	//kernel code
	__global__ void CompareVectorsKernel(float* inputOne, float* inputTwo, float* output)
	{
		int id = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
			+ blockDim.x*blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;
		
		if (inputOne[id] != inputTwo[id])
			output[0] = 1;
	}
}
