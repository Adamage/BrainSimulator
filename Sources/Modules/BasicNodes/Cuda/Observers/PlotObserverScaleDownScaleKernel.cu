#include "hip/hip_runtime.h"
#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

#include <hip/hip_runtime.h> 
#include <hip/hip_math_constants.h> 
#include <hip/hip_runtime_api.h>
#include <> 
#include <> 
#include <builtin_types.h> 
#include <hip/hip_vector_types.h> 
#include <vector_functions.h> 



extern "C"  
{	
	//kernel code
	__global__ void PlotObserverScaleDownScaleKernel(float* history, int nbCurves, int size)
	{		
		int id = blockDim.x*blockIdx.y*gridDim.x	
				+ blockDim.x*blockIdx.x				
				+ threadIdx.x;

		if (id >= size)
			return;
		
		int baseAddress = 2 * id;
		float val1 = history[baseAddress];
		float val2 = history[baseAddress + nbCurves];
		float average = (val1 + val2) / 2;
		history[id] = average;
	}
}