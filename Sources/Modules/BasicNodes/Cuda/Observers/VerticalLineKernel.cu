#include "hip/hip_runtime.h"
#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

#include <hip/hip_runtime.h> 
#include <hip/hip_math_constants.h> 
#include <hip/hip_runtime_api.h>
#include <> 
#include <> 
#include <builtin_types.h> 
#include <hip/hip_vector_types.h> 
#include <vector_functions.h> 


#define COLOR_GREY	0xFF888888

extern "C"  
{	

	//kernel code
	__global__ void VerticalLineKernel(unsigned int* canvas, int column, int width, int height)
	{		
		int id = blockDim.x*blockIdx.y*gridDim.x	
				+ blockDim.x*blockIdx.x				
				+ threadIdx.x;

		if (id >= height)
			return;
		
		canvas[column + id * width] = COLOR_GREY;	
	}
}