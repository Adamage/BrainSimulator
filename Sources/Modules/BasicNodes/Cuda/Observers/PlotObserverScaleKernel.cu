#include "hip/hip_runtime.h"
#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

#include <hip/hip_runtime.h> 
#include <hip/hip_math_constants.h> 
#include <hip/hip_runtime_api.h>
#include <> 
#include <> 
#include <builtin_types.h> 
#include <hip/hip_vector_types.h> 
#include <vector_functions.h> 


extern "C"  
{	
	__constant__ int D_NB_CURVES;
	__constant__ int D_TEXTURE_WIDTH;
	__constant__ int D_PLOTAREA_WIDTH;
	__constant__ int D_PLOTAREA_HEIGHT;
	__constant__ int D_PLOTAREA_OFFSET_X;
	__constant__ double D_MIN_VALUE;
	__constant__ double D_MAX_VALUE;
	__constant__ unsigned int D_COLOR_BACKGROUND;
	__constant__ unsigned int D_COLOR_CURVES[6];
	__constant__ unsigned int D_COLOR_CURVE_EXTRA;

	//kernel code
	__global__ void PlotObserverScaleKernel(unsigned int* canvas, int columnStart, int nbColumn, float* values)
	{		
		int id = blockDim.x*blockIdx.y*gridDim.x	
				+ blockDim.x*blockIdx.x				
				+ threadIdx.x;

		if (id >= nbColumn * D_PLOTAREA_HEIGHT)
			return;
		
		int x = (columnStart + id % nbColumn) % D_PLOTAREA_WIDTH; 
		int y = id / nbColumn;


		double valueRange = D_MAX_VALUE - D_MIN_VALUE;
		double pixelRange = valueRange / D_PLOTAREA_HEIGHT;
		
		double displayWindowMinValue = y * pixelRange;
		double displayWindowMaxValue = displayWindowMinValue + pixelRange;


		
		// For each curve
		unsigned int color = D_COLOR_BACKGROUND;
		for (int c = 0; c < D_NB_CURVES; c++)
		{
			double currentValue = values[x * D_NB_CURVES + c];
			double previousValue = (x > 0) ? values[(x - 1) * D_NB_CURVES + c] : currentValue;
			
			double currentRelativeValue = currentValue - D_MIN_VALUE; // Use a relative value to keep the high precision if the value is very high
			double previousRelativeValue = previousValue - D_MIN_VALUE;

			if (	currentRelativeValue >= displayWindowMinValue && currentRelativeValue < displayWindowMaxValue
				|| previousRelativeValue < displayWindowMinValue && currentRelativeValue > displayWindowMaxValue
				|| previousRelativeValue > displayWindowMaxValue && currentRelativeValue < displayWindowMinValue
				)
			{
				color = (c < 6) ? D_COLOR_CURVES[c] : D_COLOR_CURVE_EXTRA;
				break;
			}
		}

		canvas[D_PLOTAREA_OFFSET_X + x + (D_PLOTAREA_HEIGHT - 1 - y) * D_TEXTURE_WIDTH] = color;	
	}
}