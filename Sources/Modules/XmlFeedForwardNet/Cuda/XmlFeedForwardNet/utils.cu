#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

#include <hip/hip_runtime.h> 
#include <hip/hip_math_constants.h> 
#include <> 
#include <> 
#include <builtin_types.h> 
#include <vector_functions.h> 
#include <float.h>


extern "C"  
{
	typedef unsigned int uint;

	struct MyLayerDim
	{
		float* Ptr;     // GPU pointer to the offseted data
        size_t Nb;      // Number of images
        size_t Width;   // Width of each image
        size_t Height;  // Height of each image
        size_t Depth;  // Depth (Used for weights)
        size_t Size;    // Size of each image (Width * Height)
        size_t Count;   // Total number of channels (Nb * Size)
	};

	
	struct KernelDim
    {
        size_t Nb;
        size_t Width;
        size_t Height;
        size_t Depth;
        size_t Size;
        size_t Count;
    };

}